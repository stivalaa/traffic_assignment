#include "hip/hip_runtime.h"
/*****************************************************************************
 * 
 * File:    pape_cuda_host.cu
 * Author:  Alex Stivala
 * Created: February 2011
 *
 * $Id: pape_cuda_host.cu 222 2011-04-13 04:01:26Z astivala $
 *
 * CUDA host code for CUDA implemnetatnion of d'Esopo-Pape algorithm.
 *
 ****************************************************************************/

#include <assert.h>

#include <cutil_inline.h>      /* CUDA SDK */

#include "sssp.h"
#include "pape_kernels.h"
#include "pape_cuda_host.h"

#define TIMER_DEBUG

//ceil(a / b)
extern "C" int iDivUp(int a, int b){
  return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

/*
 * pape_cuda() - multiple-source shortest path by d'Esopo-Pape algorhtm.
 *
 * Parameters:
 *    Va, Ea, Wa - graph in packed adjacency list represention
 *    num_nodes - number of nodes (elemnts in  Va)
 *    num_edges - number of edges (elements in Ea, Wa)
 *    start_nodes - array of source nodes
 *    num_start_nodes - number of source nodes (elements in start_nodes)
 *    distances (OUT) - 2d array of shortest costs from sources to each node
 *                      distances[i*num_nodes+j] is cost from sourc j to node i
 *    predecessors (OUT) - 2d array (as above)
 *                        of predecessor nodes for each node
 *
 * Return value:
 *    None.
 *
 * Each CUDA thread does one source node.
 *
 */
void pape_cuda(int Va[], int Ea[], float Wa[], 
               int num_nodes, int num_edges,
               int start_nodes[], int num_start_nodes,
               float *distances, int *predecessors)
{
  int *d_Va = NULL, *d_Ea;
  float *d_Wa;
  int *d_start_nodes;
  int *d_Pa;
  float *d_Ca;
  unsigned int hTimer;
  double copytime, runtime;
  int first_thru_node = 0;
  int *d_queue_next;

  // allocate arrays for packed adjancey list format and 
  // copy graph in packed adjacney list format to device
  // also start nodes list
  cutilSafeCall( hipMalloc((void **)&d_Va, (num_nodes+1)*sizeof(int)) );
  cutilSafeCall( hipMalloc((void **)&d_Ea, num_edges*sizeof(int)) );
  cutilSafeCall( hipMalloc((void **)&d_Wa, num_edges*sizeof(float)) );
  cutilSafeCall( hipMalloc((void **)&d_start_nodes, num_start_nodes*sizeof(int)) );

  printf("%d nodes (%d KB) %d edges (%d KB)\n",
         num_nodes, 
         ( (num_nodes+1)*sizeof(int) ) / 1024,
         num_edges,
         ( num_edges*sizeof(int) + num_edges*sizeof(float) ) / 1024);

  cutilCheckError( cutCreateTimer(&hTimer) );
  cutilCheckError( cutResetTimer(hTimer) );
  cutilCheckError( cutStartTimer(hTimer) );

  cutilSafeCall( hipMemcpy(d_Va, Va, (num_nodes+1)*sizeof(int),
                            hipMemcpyHostToDevice) );
  cutilSafeCall( hipMemcpy(d_Ea, Ea, num_edges*sizeof(int),
                            hipMemcpyHostToDevice) );
  cutilSafeCall( hipMemcpy(d_Wa, Wa, num_edges*sizeof(float),
                            hipMemcpyHostToDevice) );
  cutilSafeCall( hipMemcpy(d_start_nodes, start_nodes, num_start_nodes*sizeof(int),
                            hipMemcpyHostToDevice) );
  
  cutilCheckError( cutStopTimer(hTimer) );
  copytime = cutGetTimerValue(hTimer);
  printf("time to copy %d nodes %d edges (total %d KB) to device: %f ms\n",
         num_nodes, num_edges,
         ( (num_nodes+1)*sizeof(int) + num_edges*sizeof(int) +
           num_edges*sizeof(float) ) / 1024,
         copytime);

  // allocate arrays for costs, predecessors, and queues
  cutilSafeCall( hipMalloc((void **)&d_Ca, num_nodes*num_start_nodes*
                            sizeof(float)) );
  cutilSafeCall( hipMalloc((void**)&d_Pa, num_nodes*num_start_nodes*
                            sizeof(int)) );
  cutilSafeCall( hipMalloc((void **)&d_queue_next, 
                            num_nodes*num_start_nodes*sizeof(int)) );

  // initialize the  modification set, cost, updated cost arrays on device
  dim3 dimBlock(num_start_nodes);    // threads per block
  dim3 dimGrid(num_nodes);           // blocks (per grid)
  if (num_start_nodes > 512) // FIXME some rule for this
    dimBlock = dim3(512); // kernel will handle multiple start nodes per thread
   if (num_nodes > 65535) // FIXME some rule for this - 65535 is Fermi max */
     dimGrid = dim3(65535);
  fprintf(stdout, "Initalize Execution configuration: Grid = (%d,%d,%d) Block = (%d,%d,%d)\n", dimGrid.x,dimGrid.y,dimGrid.z, dimBlock.x,dimBlock.y,dimBlock.z);
#ifdef TIMER_DEBUG
    unsigned int tdhTimer;
    cutilCheckError( cutCreateTimer(&tdhTimer) );
    cutilCheckError( cutResetTimer(tdhTimer) );
    cutilCheckError( cutStartTimer(tdhTimer) );
#endif /* TIMER_DEBUG */

  pape_init_arrays<<<dimGrid, dimBlock>>>(num_nodes, num_start_nodes,
                                          d_start_nodes, d_Ca, d_Pa,
                                          d_queue_next);
  CUT_CHECK_ERROR("Kernel execution failed (okuyama_init_mask_cost_update_arrays)");
  cutilSafeCall( hipDeviceSynchronize() );

#ifdef TIMER_DEBUG
    cutilCheckError( cutStopTimer(tdhTimer) );
    double init_time = cutGetTimerValue(tdhTimer);
    fprintf(stderr, "pape_init_array time: %f ms\n", init_time);
#endif /* TIMER_DEBUG */



  dimBlock = dim3(num_start_nodes);    // threads per block
  dimGrid = dim3(1);           // blocks (per grid)
  if (num_start_nodes > 512) // FIXME some rule for this
  {
    dimBlock = dim3(512); // kernel will handle multiple start nodes per thread
    dimGrid = dim3(iDivUp(num_start_nodes, 512));
  }
  fprintf(stdout, "Execution configuration: Grid = (%d,%d,%d) Block = (%d,%d,%d)\n", dimGrid.x,dimGrid.y,dimGrid.z, dimBlock.x,dimBlock.y,dimBlock.z);

#ifdef TIMER_DEBUG
    cutilCheckError( cutCreateTimer(&tdhTimer) );
    cutilCheckError( cutResetTimer(tdhTimer) );
    cutilCheckError( cutStartTimer(tdhTimer) );
#endif /* TIMER_DEBUG */
    
    pape_kernel<<<dimGrid, dimBlock>>>(d_Va, d_Ea, d_Wa,
                                       num_nodes, num_edges, num_start_nodes,
                                       d_start_nodes, first_thru_node,
                                       d_Ca, d_Pa, d_queue_next);

    CUT_CHECK_ERROR("Kernel execution failed (pape_kernel)");
    cutilSafeCall( hipDeviceSynchronize() );
#ifdef TIMER_DEBUG
    cutilCheckError( cutStopTimer(tdhTimer) );
    double pape_time = cutGetTimerValue(tdhTimer);
    fprintf(stderr, "pape_kernel time: %f ms\n", pape_time);
#endif /* TIMER_DEBUG */

  // get the final costs and predecessor nodes back from the device
  cutilSafeCall( hipMemcpy(distances, d_Ca, 
                            num_nodes*num_start_nodes*sizeof(float),
                            hipMemcpyDeviceToHost) );
  cutilSafeCall( hipMemcpy(predecessors, d_Pa,
                            num_nodes*num_start_nodes*sizeof(int),
                            hipMemcpyDeviceToHost) );

  // free device memory
  cutilSafeCall( hipFree(d_Ca) );
  cutilSafeCall( hipFree(d_Pa) );
  cutilSafeCall( hipFree(d_start_nodes) );
  cutilSafeCall( hipFree(d_Va) );
  cutilSafeCall( hipFree(d_Ea) );
  cutilSafeCall( hipFree(d_Wa) );
  cutilSafeCall( hipFree(d_queue_next) );

}
